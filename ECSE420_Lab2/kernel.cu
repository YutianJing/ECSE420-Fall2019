#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "lodepng.h"
#include "wm.h"

#include "A_3.h"
#include "A_10.h"
#include "A_32.h"
#include "A_512.h"
#include "A_1024.h"
#include "b_3.h"
#include "b_10.h"
#include "b_32.h"
#include "b_512.h"
#include "b_1024.h"
#include "X_32.h"
#include "X_512.h"
#include "X_1024.h"

#define NUM_THREADS 8
#define wmSIZE 3

// do not set matrixSIZE to {32, 512, 1024}, since the inverse matrix method below is defined using matrixSIZE,
// and they are using too much recursion, complexity is high, will take forever to run
#define matrixSIZE 10

float AInv[matrixSIZE][matrixSIZE];

__global__ void convolve(unsigned char* image, unsigned char* new_image, unsigned width, unsigned height, int round, float* wm_dev)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float patch[wmSIZE * wmSIZE];
	float sum;
	unsigned offset;
	//printf("%d\n", round);
	if (i < NUM_THREADS) {
		for (int k = 0; k < 4; k++) {
			offset = round * NUM_THREADS + i;
			//offset += width * (offset / width);
			//offset -= offset % (width - wmSIZE);

			if ((offset % width) < (width - wmSIZE + 1) && offset < width * (height - wmSIZE + 1)) {
				sum = 0;
				for (int j = 0; j < wmSIZE * wmSIZE; j++) {
					patch[j] = image[(offset + width * (j / wmSIZE) + (j - wmSIZE * (j / wmSIZE))) * 4 + k];

					patch[j] = patch[j] * wm_dev[j];

					sum += patch[j];
				}
				if (sum < 0.0) sum = 0;
				if (sum > 255.0) sum = 255;
				if (k == 3) sum = image[offset * 4 + k];

				new_image[(offset - (offset / width) * (wmSIZE - 1)) * 4 + k] = sum;
			}
		}
	}
}

void imageConvolution(char* input_filename, char* output_filename)
{
	unsigned error;
	unsigned char* image, * new_image;
	unsigned width, height;

	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if (error) printf("error %u: %s\n", error, lodepng_error_text(error));
	new_image = (unsigned char*)malloc((width - wmSIZE + 1) * (height - wmSIZE + 1) * sizeof(unsigned char));

	clock_t start, end;
	start = clock();
	////////////////////////////////////////////////////////////
	hipSetDevice(0);

	unsigned char* image_dev;
	hipMallocManaged((void**)&image_dev, width * height * 4 * sizeof(unsigned char));
	hipMallocManaged((void**)&new_image, (width - wmSIZE + 1) * (height - wmSIZE + 1) * 4 * sizeof(unsigned char));
	for (int i = 0; i < width * height * 4; i++) {
		image_dev[i] = image[i];
	}
	for (int i = 0; i < (width - wmSIZE + 1) * (height - wmSIZE + 1) * 4; i++) new_image[i] = 0;
	
	float* wm_dev;
	hipMallocManaged((void**)&wm_dev, wmSIZE * wmSIZE * sizeof(float));
	for (int i = 0; i < wmSIZE; i++) {
		for (int j = 0; j < wmSIZE; j++) {
			if (wmSIZE == 3) wm_dev[i * wmSIZE + j] = w3[i][j];
			if (wmSIZE == 5) wm_dev[i * wmSIZE + j] = w5[i][j];
			if (wmSIZE == 7) wm_dev[i * wmSIZE + j] = w7[i][j];
		}
	}

	int round = 0;
	int numBlocks = (int)ceil(((double)NUM_THREADS + (double)1023) / (double)1024);
	while (round < width * height / NUM_THREADS) {
		convolve << <numBlocks, 1024 >> > (image_dev, new_image, width, height, round, wm_dev);
		round++;
	}
	//convolve << <numBlocks, (height * width) % 1024 >> > (image_dev, new_image, width, height, round, wm_dev);

	hipDeviceSynchronize();
	////////////////////////////////////////////////////////////
	end = clock();
	printf("imageConvolution time spent = %f\n", (double)(end - start) / (double)CLOCKS_PER_SEC);
	
	lodepng_encode32_file(output_filename, new_image, (width - wmSIZE + 1), (height - wmSIZE + 1));
	hipFree(image); hipFree(new_image); hipFree(image_dev); hipFree(wm_dev);
	free(image);
	//free(new_image);
	//free(image_dev);
}

double getDet(double inputMatrix[matrixSIZE][matrixSIZE], int n)
{
	if (n == 1) return inputMatrix[0][0];
	double ans = 0;
	double temp[matrixSIZE][matrixSIZE];
	int i, j, k;
	for (i = 0; i < n; i++) {
		for (j = 0; j < n - 1; j++) {
			for (k = 0; k < n - 1; k++) {
				temp[j][k] = inputMatrix[j + 1][(k >= i) ? k + 1 : k];
			}
		}
		double t = getDet(temp, n - 1);
		if (i % 2 == 0) {
			ans += inputMatrix[0][i] * t;
		}
		else {
			ans -= inputMatrix[0][i] * t;
		}
	}
	return ans;
}

void getAStar(double inputMatrix[matrixSIZE][matrixSIZE], int n, double ans[matrixSIZE][matrixSIZE])
{
	if (n == 1) {
		ans[0][0] = 1;
		return;
	}
	int i, j, k, t;
	double temp[matrixSIZE][matrixSIZE];
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			for (k = 0; k < n - 1; k++) {
				for (t = 0; t < n - 1; t++) {
					temp[k][t] = inputMatrix[k >= i ? k + 1 : k][t >= j ? t + 1 : t];
				}
			}
			ans[j][i] = getDet(temp, n - 1);
			if ((i + j) % 2 == 1) {
				ans[j][i] = -ans[j][i];
			}
		}
	}
}

double inverse(double inputMatrix[matrixSIZE][matrixSIZE])
{
	double AStar[matrixSIZE][matrixSIZE];
	double det = getDet(inputMatrix, matrixSIZE);
	if (det == 0) { printf("The input matrix can not be transformed!\n"); }
	else {
		getAStar(inputMatrix, matrixSIZE, AStar);
		for (int i = 0; i < matrixSIZE; i++) {
			for (int j = 0; j < matrixSIZE; j++) {
				AInv[i][j] = AStar[i][j] / det;
			}
		}
	}
	return 0;
}

__global__ void multiply_and_add(float* x, float* AInv_dev, float* b_dev, int round)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned offset;
	if (i < NUM_THREADS) {
		offset = round * NUM_THREADS + i;
		for (int j = 0; j < matrixSIZE; j++) {
			x[offset] += AInv_dev[offset * matrixSIZE + j % matrixSIZE] * b_dev[j];
		}
		printf("x[%d] = %f\n", offset, x[offset]);
	}
}

void solve_Ax_equals_b()
{	
	////////////////////////////////////////////////// Question 2: Solve Ax = b
	double A[matrixSIZE][matrixSIZE];
	for (int i = 0; i < matrixSIZE; i++) {
		for (int j = 0; j < matrixSIZE; j++) {
			if (matrixSIZE == 3) A[i][j] = A_3[i][j];
			if (matrixSIZE == 10) A[i][j] = A_10[i][j];
			if (matrixSIZE == 32) A[i][j] = A_32[i][j];
			if (matrixSIZE == 512) A[i][j] = A_512[i][j];
			if (matrixSIZE == 1024) A[i][j] = A_1024[i][j];
		}
	}

	// inverse of a 32x32 matrix takes too long
	inverse(A); // now the inverse is stored in AInv

	hipSetDevice(0);

	float* x;
	x = (float*)malloc(matrixSIZE * sizeof(float));
	hipMallocManaged((void**)&x, matrixSIZE * sizeof(float));
	for (int i = 0; i < matrixSIZE; i++) {
		if (matrixSIZE == 32) x[i] = X_32[i][0];
		if (matrixSIZE == 512) x[i] = X_512[i][0];
		if (matrixSIZE == 1024) x[i] = X_1024[i][0];
		else x[i] = 0;
	}

	float* AInv_dev;
	hipMallocManaged((void**)&AInv_dev, matrixSIZE * matrixSIZE * sizeof(float));
	for (int i = 0; i < matrixSIZE; i++) {
		for (int j = 0; j < matrixSIZE; j++) {
			AInv_dev[i * matrixSIZE + j] = AInv[i][j];
		}
	}

	float* b_dev;
	hipMallocManaged((void**)&b_dev, matrixSIZE * sizeof(float));
	for (int i = 0; i < matrixSIZE; i++) {
		if (matrixSIZE == 3) b_dev[i] = b_3[i][0];
		if (matrixSIZE == 10) b_dev[i] = b_10[i][0];
		if (matrixSIZE == 32) b_dev[i] = b_32[i][0];
		if (matrixSIZE == 512) b_dev[i] = b_512[i][0];
		if (matrixSIZE == 1024) b_dev[i] = b_1024[i][0];
	}
	
	int round = 0;
	while (round < (int)ceil((double)matrixSIZE / (double)NUM_THREADS)) {
		multiply_and_add << <1, matrixSIZE >> > (x, AInv_dev, b_dev, round);
		round++;
	}
	
	hipDeviceSynchronize();

	float A_times_x[matrixSIZE];
	for (int i = 0; i < matrixSIZE; i++) {
		A_times_x[i] = 0;
		for (int j = 0; j < matrixSIZE; j++) {
			if (matrixSIZE == 32) A_times_x[i] += A[i][j] * X_32[j][0];
			if (matrixSIZE == 512) A_times_x[i] += A[i][j] * X_512[j][0];
			if (matrixSIZE == 1024) A_times_x[i] += A[i][j] * X_1024[j][0];
			else A_times_x[i] += A[i][j] * x[j];
		}
	}
	for (int i = 0; i < matrixSIZE; i++) {
		printf("A[%d][:] * x[:] - b[%d] = %f\n", i, i, A_times_x[i] - b_dev[i]);
	}

	hipFree(AInv_dev); hipFree(b_dev); hipFree(x);
}

int main()
{
	char* input_filename = "test.png";
	char* output_filename_convolution3 = "test convolve 3x3.png";
	char* output_filename_convolution5 = "test convolve 5x5.png";
	char* output_filename_convolution7 = "test convolve 7x7.png";

	if (wmSIZE == 3) imageConvolution(input_filename, output_filename_convolution3);
	if (wmSIZE == 5) imageConvolution(input_filename, output_filename_convolution5);
	if (wmSIZE == 7) imageConvolution(input_filename, output_filename_convolution7);
	
	solve_Ax_equals_b();

	return 0;
}
