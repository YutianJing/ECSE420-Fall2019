#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "lodepng.h"
#include "wm.h"
#include "A_32.h"
#include "A_512.h"
#include "A_1024.h"
#include "b_32.h"
#include "b_512.h"
#include "b_1024.h"
#define NUM_THREADS 1024
#define wmDimension 3

__global__ void convolve(unsigned char* image, unsigned char* new_image, unsigned width, unsigned height, int round, float* wm_dev)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float patch[wmDimension * wmDimension];
	float sum;
	unsigned offset;
	//printf("%d\n", round);
	if (i < NUM_THREADS) {
		for (int k = 0; k < 4; k++) {
			offset = round * NUM_THREADS + i;
			//offset += width * (offset / width);
			
			if (offset % width < (width - wmDimension + 1) && offset < width * (height - wmDimension + 1)) {
				sum = 0;
				for (int j = 0; j < wmDimension * wmDimension; j++) {
					patch[j] = image[(offset + width * (j / wmDimension) + (j - wmDimension * (j / wmDimension))) * 4 + k];

					patch[j] = patch[j] * wm_dev[j];

					sum += patch[j];
				}
				if (sum < 0.0) sum = 0;
				if (sum > 255.0) sum = 255;
				if (k == 3) sum = 255;

				new_image[offset * 4 + k] = sum;
			}
		}
	}
}

void imageConvolution(char* input_filename, char* output_filename)
{
	unsigned error;
	unsigned char* image, * new_image;
	unsigned width, height;

	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if (error) printf("error %u: %s\n", error, lodepng_error_text(error));
	new_image = (unsigned char*)malloc((width - wmDimension + 1) * (height - wmDimension + 1) * sizeof(unsigned char));

	clock_t start, end;
	start = clock();
	////////////////////////////////////////////////////////////////////////////////
	hipSetDevice(0);

	unsigned char* image_dev;
	hipMallocManaged((void**)&image_dev, width * height * 4 * sizeof(unsigned char));
	hipMallocManaged((void**)&new_image, (width - wmDimension + 1) * (height - wmDimension + 1) * 4 * sizeof(unsigned char));
	for (int i = 0; i < width * height * 4; i++) {
		image_dev[i] = image[i];
	}
	for (int i = 0; i < (width - wmDimension + 1) * (height - wmDimension + 1) * 4; i++) new_image[i] = 0;
	
	float* wm_dev;
	hipMallocManaged((void**)&wm_dev, wmDimension * wmDimension * sizeof(float));
	for (int i = 0; i < wmDimension; i++) {
		for (int j = 0; j < wmDimension; j++) {
			if (wmDimension == 3) wm_dev[i * wmDimension + j] = w3[i][j];
			if (wmDimension == 5) wm_dev[i * wmDimension + j] = w5[i][j];
			if (wmDimension == 7) wm_dev[i * wmDimension + j] = w7[i][j];
		}
	}

	int round = 0;
	int numBlocks = (int)ceil(((double)NUM_THREADS + (double)1023) / (double)1024);
	while (round < (width - wmDimension + 1) * (height - wmDimension + 1) / NUM_THREADS) {
		convolve << <numBlocks, 1024 >> > (image_dev, new_image, width, height, round, wm_dev);
		round++;
	}
	//convolve << <numBlocks, (height * width) % 1024 >> > (image_dev, new_image, width, height, round, w3, w5, w7);

	hipDeviceSynchronize();
	////////////////////////////////////////////////////////////////////////////////
	end = clock();
	printf("time=%f\n", (double)(end - start) / (double)CLOCKS_PER_SEC);
	
	lodepng_encode32_file(output_filename, new_image, (width - wmDimension + 1), (height - wmDimension + 1));
	hipFree(image); hipFree(new_image); hipFree(image_dev); hipFree(wm_dev);
	free(image);
	//free(new_image);
	//free(image_dev);
}

int main()
{

	char* input_filename = "test.png";
	char* output_filename_convolution3 = "test convolve 3x3.png";
	char* output_filename_convolution5 = "test convolve 5x5.png";
	char* output_filename_convolution7 = "test convolve 7x7.png";

	if (wmDimension == 3) imageConvolution(input_filename, output_filename_convolution3);
	if (wmDimension == 5) imageConvolution(input_filename, output_filename_convolution5);
	if (wmDimension == 7) imageConvolution(input_filename, output_filename_convolution7);
	
	return 0;

}
