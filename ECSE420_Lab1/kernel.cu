#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "lodepng.h"
#define NUM_THREADS 256
__global__ void rectify(unsigned char* image, unsigned char* new_image, int round, int numThreads)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//int magicNumber = (numThreads / 1024) * 1024;
	int magicNumber = NUM_THREADS;

	if (i < numThreads) {
		
		if (image[(round * magicNumber + i) * 4] >= 127) // R
			new_image[(round * magicNumber + i) * 4] = image[(round * magicNumber + i) * 4];
		else new_image[(round * magicNumber + i) * 4] = 127;

		if (image[(round * magicNumber + i) * 4 + 1] >= 127) // G
			new_image[(round * magicNumber + i) * 4 + 1] = image[(round * magicNumber + i) * 4 + 1];
		else new_image[(round * magicNumber + i) * 4 + 1] = 127;

		if (image[(round * magicNumber + i) * 4 + 2] >= 127) // B
			new_image[(round * magicNumber + i) * 4 + 2] = image[(round * magicNumber + i) * 4 + 2];
		else new_image[(round * magicNumber + i) * 4 + 2] = 127;

		new_image[(round * magicNumber + i) * 4 + 3] = image[(round * magicNumber + i) * 4 + 3]; // A
	}
}

__global__ void pool(unsigned char* image, unsigned char* new_image, unsigned width, unsigned height, int round, int numThreads)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//int magicNumber = (numThreads / 1024) * 1024;
	int magicNumber = NUM_THREADS;
	unsigned char tl, tr, bl, br, max;
	unsigned offset;

	if (i < numThreads) {
		for (int k = 0; k < 4; k++) {
			offset = round * magicNumber * 2 + i * 2;
			offset += width * (offset / width);

			tl = image[(offset) * 4 + k];
			tr = image[(offset + 1) * 4 + k];
			bl = image[(offset + width) * 4 + k];
			br = image[(offset + width + 1) * 4 + k];

			max = 0;

			if (tl > max) max = tl;
			if (tr > max) max = tr;
			if (bl > max) max = bl;
			if (br > max) max = br;

			new_image[(round * magicNumber + i) * 4 + k] = max;
		}
	}
}

void imageRectify(char* input_filename, char* output_filename)
{
	unsigned error;
	unsigned char* image, * new_image;
	unsigned width, height;

	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if (error) printf("error %u: %s\n", error, lodepng_error_text(error));
	new_image = (unsigned char*) malloc(width * height * 4 * sizeof(unsigned char));

	// sequential way of rectifying
	//for (int i = 0; i < height; i++) {
	//	for (int j = 0; j < width; j++) {

	//		if (image[4 * width * i + 4 * j] >= 127)
	//			new_image[4 * width * i + 4 * j] = image[4 * width * i + 4 * j]; // R
	//		else new_image[4 * width * i + 4 * j] = 127;

	//		if (image[4 * width * i + 4 * j + 1] >= 127)
	//			new_image[4 * width * i + 4 * j + 1] = image[4 * width * i + 4 * j + 1]; // G
	//		else new_image[4 * width * i + 4 * j + 1] = 127;

	//		if (image[4 * width * i + 4 * j + 2] >= 127)
	//			new_image[4 * width * i + 4 * j + 2] = image[4 * width * i + 4 * j + 2]; // B
	//		else new_image[4 * width * i + 4 * j + 2] = 127;

	//		new_image[4 * width * i + 4 * j + 3] = image[4 * width * i + 4 * j + 3]; // A

	//	}
	//}
	//clock_t start, end;
	//start = clock();
	////////////////////////////////////////////////////////////////////////////////
	// parallel way of rectifying
	hipSetDevice(0);

	unsigned char* image_dev;
	hipMallocManaged((void**)&image_dev, width * height * 4 * sizeof(unsigned char));
	hipMallocManaged((void**)&new_image, width * height * 4 * sizeof(unsigned char));
	for (int i = 0; i < width * height * 4; i++) {
		image_dev[i] = image[i];
		new_image[i] = 0;
	}

	int round = 0;
	while (round < width * height / NUM_THREADS) {
		rectify << <(int)ceil((NUM_THREADS + 1023) / 1024), 1024 >> > (image_dev, new_image, round, NUM_THREADS);
		round++;
	}
		rectify << <(int)ceil((NUM_THREADS + 1023) / 1024), (height * width) % 1024 >> > (image_dev, new_image, round, NUM_THREADS);

	hipDeviceSynchronize();
	//hipFree(image); hipFree(new_image); hipFree(width_p); hipFree(height_p); hipFree(image_dev); hipFree(new_image_dev);
	//end = clock();
	//printf("time=%f\n", (double)(end - start) / (double)CLOCKS_PER_SEC);
	//////////////////////////////////////////////////////////////////////////////////

	lodepng_encode32_file(output_filename, new_image, width, height);
	hipFree(image); hipFree(new_image); hipFree(image_dev);
	free(image);
	//free(new_image);
	//free(image_dev);
}

void imagePooling(char* input_filename, char* output_filename)
{
	unsigned error;
	unsigned char* image, * new_image;
	unsigned width, height;

	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if (error) printf("error %u: %s\n", error, lodepng_error_text(error));
	new_image = (unsigned char*)malloc(width * height * sizeof(unsigned char));

	clock_t start, end;
	start = clock();
	////////////////////////////////////////////////////////////////////////////////
	// parallel way of pooling
	hipSetDevice(0);

	unsigned char* image_dev;
	hipMallocManaged((void**)&image_dev, width * height * 4 * sizeof(unsigned char));
	hipMallocManaged((void**)&new_image, width * height * sizeof(unsigned char));
	for (int i = 0; i < width * height * 4; i++) {
		image_dev[i] = image[i];
	}
	for (int i = 0; i < width * height; i++) new_image[i] = 0;

	int round = 0;
	while (round < width * height / NUM_THREADS / 4) {
		pool << <(int)ceil((NUM_THREADS + 1023) / 1024), 1024 >> > (image_dev, new_image, width, height, round, NUM_THREADS);
		round++;
	}
		pool << <(int)ceil((NUM_THREADS + 1023) / 1024), (height * width) % 1024 >> > (image_dev, new_image, width, height, round, NUM_THREADS);

	hipDeviceSynchronize();
	end = clock();
	printf("time=%f\n", (double)(end - start) / (double)CLOCKS_PER_SEC);
	//////////////////////////////////////////////////////////////////////////////////

	lodepng_encode32_file(output_filename, new_image, width / 2, height / 2);
	hipFree(image); hipFree(new_image); hipFree(image_dev);
	free(image);
	//free(new_image);
	//free(image_dev);
}

int main()
{

	char* input_filename = "test.png";
	char* output_filename_rectify = "test rectify.png";
	char* output_filename_pooling = "test pooling.png";

	imageRectify(input_filename, output_filename_rectify);
	imagePooling(input_filename, output_filename_pooling);

	return 0;

}
